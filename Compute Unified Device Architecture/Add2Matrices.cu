/*
Write CUDA program to add 2 Matrices A and B and store result in C.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 2

__global__ void MatAdd(int A[][N], int B[][N], int C[][N])
{
    int i = threadIdx.x;
    int j = threadIdx.y;
    C[i][j] = A[i][j] + B[i][j];
}

int main()
{
    // create events
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    
    int A[N][N] = {{1, 2}, {3, 4}};
    int B[N][N] = {{5, 6}, {7, 8}};
    int C[N][N] = {{0, 0}, {0, 0}};
    
    int(*pA)[N], (*pB)[N], (*pC)[N];
    
    hipMalloc((void **)&pA, (N * N) * sizeof(int));
    hipMalloc((void **)&pB, (N * N) * sizeof(int));
    hipMalloc((void **)&pC, (N * N) * sizeof(int));
    
    hipMemcpy(pA, A, (N * N) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(pB, B, (N * N) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(pC, C, (N * N) * sizeof(int), hipMemcpyHostToDevice);
    
    int numBlocks = 1;
    
    dim3 threadsPerBlock(N, N);
    
    // record events around kernel launch
    hipEventRecord(event1, 0); // where 0 is the default stream
    
    MatAdd<<<numBlocks, threadsPerBlock>>>(pA, pB, pC);
    
    hipMemcpy(C, pC, (N * N) * sizeof(int), hipMemcpyDeviceToHost);
    
    hipEventRecord(event2, 0);
    
    int i, j;
    
    printf("C = \n");
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
            printf("%d ", C[i][j]);
        printf("\n");
    }
    
    // synchronize
    hipEventSynchronize(event1); // optional
    hipEventSynchronize(event2); // wait for the event to be executed!
    
    // calculate time
    float dt_ms;
    
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("Time: %f", dt_ms);
    
    hipFree(pA);
    hipFree(pB);
    hipFree(pC);
    
    return 0;
}